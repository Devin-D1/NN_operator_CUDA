
#include <hip/hip_runtime.h>
__global__ void tensor_add_kernel(float* c, const float* a, const float* b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

void tensor_add_ng(float* c, const float* a, const float* b, int n)
{
	dim3 grid((n + 1023) / 1024);
	dim3 block(1024);
	tensor_add_kernel<<<grid, block>>>(c, a, b);
}
